#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount failed: " << hipGetErrorString(error_id) << std::endl;
        return 1;
    }

    if (deviceCount == 0) {
        std::cout << "Not found :(" << std::endl;
    } else {
        std::cout << "Found " << deviceCount << " CUDA-enabled device(s)." << std::endl;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        std::cout << "\n--- Device " << dev << ": " << deviceProp.name << " ---" << std::endl;
        std::cout << "  Compute capability:          " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Memory:         " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Number of multiprocessors:   " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Core count:             " << deviceProp.multiProcessorCount * 192 << " (for reference, may vary)" << std::endl;
    }

    return 0;
}